#include "hip/hip_runtime.h"
#include "whisper/beam_search/logit_processor.h"
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <algorithm>
#include <stdexcept>
#include <cmath>

namespace whisper {
namespace beam_search {

// CUDA kernels

// Apply temperature to logits
__global__ void TemperatureKernel(float* logits, size_t size, float temperature) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        logits[idx] = logits[idx] / temperature;
    }
}

// Find max value for stable softmax
__global__ void LogitMaxKernel(const float* logits, float* max_values, int vocab_size) {
    int batch_pos = blockIdx.x;
    int offset = batch_pos * vocab_size;
    
    // Using shared memory for reduction
    __shared__ float shared_max[256];
    
    // Initialize with first value
    float max_val = logits[offset];
    
    // Find max across vocabulary
    for (int i = threadIdx.x; i < vocab_size; i += blockDim.x) {
        max_val = max(max_val, logits[offset + i]);
    }
    
    // Store thread's max to shared memory
    shared_max[threadIdx.x] = max_val;
    __syncthreads();
    
    // Reduce within block
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_max[threadIdx.x] = max(shared_max[threadIdx.x], shared_max[threadIdx.x + stride]);
        }
        __syncthreads();
    }
    
    // Write result
    if (threadIdx.x == 0) {
        max_values[batch_pos] = shared_max[0];
    }
}

// Compute softmax efficiently
__global__ void SoftmaxKernel(
    const float* logits, float* probs, const float* max_values, 
    int vocab_size, float* sum_values) {
    
    int batch_pos = blockIdx.x;
    int offset = batch_pos * vocab_size;
    float max_val = max_values[batch_pos];
    
    // Using shared memory for sum reduction
    __shared__ float shared_sum[256];
    shared_sum[threadIdx.x] = 0.0f;
    
    // Compute exp(logit - max) and sum
    float thread_sum = 0.0f;
    for (int i = threadIdx.x; i < vocab_size; i += blockDim.x) {
        float val = expf(logits[offset + i] - max_val);
        probs[offset + i] = val;
        thread_sum += val;
    }
    
    // Store thread's sum to shared memory
    shared_sum[threadIdx.x] = thread_sum;
    __syncthreads();
    
    // Reduce within block
    for (int stride = blockDim.x/2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Write sum result
    if (threadIdx.x == 0) {
        sum_values[batch_pos] = shared_sum[0];
    }
}

// Normalize values after softmax
__global__ void NormalizeKernel(float* probs, const float* sum_values, int vocab_size) {
    int batch_pos = blockIdx.x;
    int offset = batch_pos * vocab_size;
    float sum = sum_values[batch_pos];
    
    // Normalize each probability
    for (int i = threadIdx.x; i < vocab_size; i += blockDim.x) {
        probs[offset + i] /= sum;
    }
}

// Generate indices for sorting
__global__ void InitIndicesKernel(int* indices, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        indices[idx] = idx;
    }
}

// Score tokens kernel
__global__ void ScoreTokensKernel(
    const float* logits, const float* prev_scores, int* token_ids, int* prev_indices,
    float* new_scores, int* new_token_ids, int* new_prev_indices,
    int beam_size, int vocab_size) {
    
    int beam_idx = blockIdx.y;
    int vocab_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (beam_idx < beam_size && vocab_idx < vocab_size) {
        int out_idx = beam_idx * vocab_size + vocab_idx;
        
        // Combine scores
        new_scores[out_idx] = prev_scores[beam_idx] + logits[vocab_idx];
        new_token_ids[out_idx] = vocab_idx;
        new_prev_indices[out_idx] = beam_idx;
    }
}

// Top-K comparator for sorting
struct ScoreComparator {
    const float* scores;
    
    ScoreComparator(const float* s) : scores(s) {}
    
    __host__ __device__ bool operator()(int a, int b) const {
        return scores[a] > scores[b]; // Descending order
    }
};

// Constants
constexpr int kBlockSize = 256;
constexpr size_t kAlignment = 128; // Align to 128 bytes

// Implementation

LogitProcessor::LogitProcessor(
    BeamSearchWorkspace* workspace, float temperature, int top_k, float top_p)
    : workspace_(workspace), temperature_(temperature), top_k_(top_k), top_p_(top_p) {
    
    if (!workspace_) {
        throw std::invalid_argument("Workspace cannot be null");
    }
}

bool LogitProcessor::ProcessLogits(
    const float* d_logits, int batch_size, int seq_len, int vocab_size) {
    
    // Store dimensions
    batch_size_ = batch_size;
    seq_len_ = seq_len;
    vocab_size_ = vocab_size;
    
    // Calculate required storage size
    size_t logits_size = batch_size * seq_len * vocab_size * sizeof(float);
    size_t indices_size = batch_size * seq_len * vocab_size * sizeof(int);
    size_t temp_size = batch_size * seq_len * sizeof(float) * 2; // For max and sum values
    
    // Total storage needed
    size_t total_size = logits_size + indices_size + temp_size;
    
    // Allocate memory if needed
    AllocateMemory(total_size);
    
    // Copy logits to our processed buffer
    hipMemcpy(d_processed_logits_, d_logits, logits_size, hipMemcpyDeviceToDevice);
    
    return true;
}

void LogitProcessor::AllocateMemory(size_t required_size) {
    // Only reallocate if needed
    if (required_size <= temp_storage_size_ && d_processed_logits_ != nullptr) {
        return;
    }
    
    // Allocate with proper alignment
    temp_storage_size_ = required_size + kAlignment;
    d_processed_logits_ = static_cast<float*>(workspace_->Allocate(
        batch_size_ * seq_len_ * vocab_size_ * sizeof(float), kAlignment));
    
    d_token_indices_ = static_cast<int*>(workspace_->Allocate(
        batch_size_ * seq_len_ * vocab_size_ * sizeof(int), kAlignment));
    
    d_temp_storage_ = static_cast<float*>(workspace_->Allocate(
        batch_size_ * seq_len_ * sizeof(float) * 2, kAlignment));
    
    if (!d_processed_logits_ || !d_token_indices_ || !d_temp_storage_) {
        throw std::runtime_error("Failed to allocate device memory for LogitProcessor");
    }
}

void LogitProcessor::ApplyTemperature(float* d_logits, int batch_index, int position) {
    int offset = (batch_index * seq_len_ + position) * vocab_size_;
    int blocks = (vocab_size_ + kBlockSize - 1) / kBlockSize;
    
    TemperatureKernel<<<blocks, kBlockSize>>>(
        d_logits + offset, vocab_size_, temperature_);
}

void LogitProcessor::ApplySoftmax(float* d_logits, int batch_index, int position) {
    int batch_pos = batch_index * seq_len_ + position;
    float* d_max_values = d_temp_storage_;
    float* d_sum_values = d_temp_storage_ + batch_size_ * seq_len_;
    
    // Find max for numerical stability
    LogitMaxKernel<<<1, kBlockSize>>>(
        d_logits + batch_pos * vocab_size_, d_max_values + batch_pos, vocab_size_);
    
    // Compute softmax
    SoftmaxKernel<<<1, kBlockSize>>>(
        d_logits + batch_pos * vocab_size_, 
        d_logits + batch_pos * vocab_size_,
        d_max_values + batch_pos, 
        vocab_size_,
        d_sum_values + batch_pos);
    
    // Normalize
    NormalizeKernel<<<1, kBlockSize>>>(
        d_logits + batch_pos * vocab_size_, 
        d_sum_values + batch_pos, 
        vocab_size_);
}

void LogitProcessor::ApplyTopK(int batch_index, int position) {
    if (top_k_ <= 0 || top_k_ >= vocab_size_) {
        return; // No need to apply top-k
    }
    
    int batch_pos = batch_index * seq_len_ + position;
    int offset = batch_pos * vocab_size_;
    int blocks = (vocab_size_ + kBlockSize - 1) / kBlockSize;
    
    // Initialize indices
    InitIndicesKernel<<<blocks, kBlockSize>>>(
        d_token_indices_ + offset, vocab_size_);
    
    // Sort indices by score
    thrust::device_ptr<int> d_indices_ptr(d_token_indices_ + offset);
    thrust::sort(
        thrust::device, 
        d_indices_ptr, 
        d_indices_ptr + vocab_size_,
        ScoreComparator(d_processed_logits_ + offset));
    
    // Only keep top-k logits (set others to -INFINITY)
    float neg_inf = -std::numeric_limits<float>::infinity();
    
    // To keep implementation simple for now, we'll do this on CPU
    // A more optimized version would use a custom kernel
    std::vector<int> h_indices(vocab_size_);
    hipMemcpy(h_indices.data(), d_token_indices_ + offset, 
              vocab_size_ * sizeof(int), hipMemcpyDeviceToHost);
    
    std::vector<float> h_logits(vocab_size_);
    hipMemcpy(h_logits.data(), d_processed_logits_ + offset, 
              vocab_size_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Set logits outside top-k to -inf
    for (int i = top_k_; i < vocab_size_; i++) {
        h_logits[h_indices[i]] = neg_inf;
    }
    
    // Copy back to device
    hipMemcpy(d_processed_logits_ + offset, h_logits.data(), 
              vocab_size_ * sizeof(float), hipMemcpyHostToDevice);
}

void LogitProcessor::ApplyTopP(int batch_index, int position) {
    if (top_p_ >= 1.0f - 1e-6) {
        return; // No need to apply top-p
    }
    
    // Basic top-p implementation
    // For brevity, we'll just do this on the CPU for now
    // A more optimized version would use custom CUDA kernels
    
    int batch_pos = batch_index * seq_len_ + position;
    int offset = batch_pos * vocab_size_;
    
    // Get sorted indices and probs
    std::vector<int> h_indices(vocab_size_);
    std::vector<float> h_probs(vocab_size_);
    
    hipMemcpy(h_indices.data(), d_token_indices_ + offset, 
              vocab_size_ * sizeof(int), hipMemcpyDeviceToHost);
    
    hipMemcpy(h_probs.data(), d_processed_logits_ + offset, 
              vocab_size_ * sizeof(float), hipMemcpyDeviceToHost);
    
    // Calculate cumulative probability
    float cum_prob = 0.0f;
    float neg_inf = -std::numeric_limits<float>::infinity();
    int cutoff_idx = vocab_size_ - 1;
    
    for (int i = 0; i < vocab_size_; i++) {
        cum_prob += h_probs[h_indices[i]];
        if (cum_prob > top_p_) {
            cutoff_idx = i;
            break;
        }
    }
    
    // Set probabilities outside top-p to 0
    for (int i = cutoff_idx + 1; i < vocab_size_; i++) {
        h_probs[h_indices[i]] = neg_inf;
    }
    
    // Copy back to device
    hipMemcpy(d_processed_logits_ + offset, h_probs.data(), 
              vocab_size_ * sizeof(float), hipMemcpyHostToDevice);
}

void LogitProcessor::ScoreNextTokens(
    const BeamArray* beam, int batch_index, int position, BeamArray* output_beam) {
    
    // Process logits for this position
    int batch_pos = batch_index * seq_len_ + position;
    int offset = batch_pos * vocab_size_;
    
    // Apply temperature and softmax
    ApplyTemperature(d_processed_logits_, batch_index, position);
    ApplySoftmax(d_processed_logits_, batch_index, position);
    
    // Apply top-k and top-p sampling if enabled
    ApplyTopK(batch_index, position);
    ApplyTopP(batch_index, position);
    
    // Now score tokens based on beam and processed logits
    size_t beam_size = beam->Size();
    size_t output_size = beam_size * vocab_size_;
    
    // Allocate temporary memory for expanded tokens
    float* d_expanded_scores = static_cast<float*>(workspace_->Allocate(
        output_size * sizeof(float), kAlignment));
    int* d_expanded_token_ids = static_cast<int*>(workspace_->Allocate(
        output_size * sizeof(int), kAlignment));
    int* d_expanded_prev_indices = static_cast<int*>(workspace_->Allocate(
        output_size * sizeof(int), kAlignment));
    
    // Get beam data pointers
    float* d_beam_scores = beam->GetScorePtr();
    int* d_beam_token_ids = beam->GetTokenIdPtr();
    int* d_beam_prev_indices = beam->GetPrevIndexPtr();
    
    // Launch kernel to score tokens
    dim3 block_dim(kBlockSize);
    dim3 grid_dim((vocab_size_ + block_dim.x - 1) / block_dim.x, beam_size);
    
    ScoreTokensKernel<<<grid_dim, block_dim>>>(
        d_processed_logits_ + offset,
        d_beam_scores, 
        d_beam_token_ids,
        d_beam_prev_indices,
        d_expanded_scores,
        d_expanded_token_ids,
        d_expanded_prev_indices,
        beam_size,
        vocab_size_);
    
    // Create expanded tokens and add to output beam
    std::vector<Token> tokens(output_size);
    
    // Copy expanded tokens to host
    std::vector<float> h_scores(output_size);
    std::vector<int> h_token_ids(output_size);
    std::vector<int> h_prev_indices(output_size);
    
    hipMemcpy(h_scores.data(), d_expanded_scores, 
              output_size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_token_ids.data(), d_expanded_token_ids, 
              output_size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_prev_indices.data(), d_expanded_prev_indices, 
              output_size * sizeof(int), hipMemcpyDeviceToHost);
    
    // Create tokens
    for (size_t i = 0; i < output_size; i++) {
        tokens[i] = Token(h_scores[i], h_token_ids[i], h_prev_indices[i]);
    }
    
    // Add expanded tokens to output beam
    output_beam->AddTokens(tokens.data(), output_size);
}

void LogitProcessor::ScoreAndPrune(
    const BeamArray* beam, int batch_index, int position, 
    BeamArray* output_beam, size_t beam_width) {
    
    // Score tokens
    ScoreNextTokens(beam, batch_index, position, output_beam);
    
    // Prune beam
    output_beam->Prune(beam_width);
}

void LogitProcessor::SetSamplingParams(float temperature, int top_k, float top_p) {
    temperature_ = temperature;
    top_k_ = top_k;
    top_p_ = top_p;
}

} // namespace beam_search
} // namespace whisper 